
#include <hip/hip_runtime.h>
#include <vector>

using Float = float;

__global__ void gpu_mul(Float * val, Float * wag, Float * inn, size_t N, size_t M)
{
	auto val_ind = blockIdx.x + blockIdx.y * N;
	auto wag_ind = blockIdx.x;
	auto inn_ind = blockIdx.y;
	
	if (val_ind < N*N and wag_ind < N and inn_ind < N) 
	{
		val[blockIdx.x + blockIdx.y * N] = wag[blockIdx.x] * inn[blockIdx.y];
	}
}


void cpu_mul(Float * val, Float * wag, Float * inn, size_t N, size_t M)
{
	Float * row_wag = wag;
	Float * row_val = val;
	for (size_t j = 0; j < M; ++j)
	{				
		for (size_t i = 0; i < N; i += 4)
		{
			row_val[i] = row_wag[i] * inn[i];	
			row_val[i+1] = row_wag[i+1] * inn[i+1];	
			row_val[i+2] = row_wag[i+2] * inn[i+2];	
			row_val[i+3] = row_wag[i+3] * inn[i+3];					
		}
		row_wag += N;
		row_val += N;
	}
}		
	

int main() 
{
	size_t N = 10000;
	size_t M = 10000;
	
	
	std::vector<Float> v;
	v.resize(N + N*M + N*M);
	Float * p = &v[0];

	Float * inn = p;
	Float * wag = p + N;
	Float * val = p+N+M*N;
		
	for (size_t i = 0; i < v.size(); ++i)
	{
		//v[i] = (int8_t)randm::uniform_f(-1, +1);
	}	

	// cpu
	if (0) {
		cpu_mul(val, wag, inn, N, M);			
	}

	// gpu
	if (1) {
	
		Float * gp;
		hipMalloc(&gp, N + N*M + N*M);
		
		Float * gpu_inn = gp;
		Float * gpu_wag = gp + N;
		Float * gpu_val = gp+N+M*N;
		
		hipMemcpy(gp, p, v.size(), hipMemcpyHostToDevice);
		
		dim3 threadsPerBlock(N,M,1);
		gpu_mul<<<1, threadsPerBlock>>>(gpu_val, gpu_wag, gpu_inn, N, M);
	
		
		
		hipMemcpy(p, gp, v.size(), hipMemcpyDeviceToHost);

		hipFree(gp);
		
	}
	
	

	return 0;
}
